#include "hip/hip_runtime.h"
/*  -*- C -*-  */
/* C interface for MARTBackProjector Kernel */
/* This file must be named .cu to be successfully compiled by nvcc */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "GPUTypes.hpp"
#include "MARTOp.Kernel.cu"

hipError_t MARTOp_GPU(bool fanbeam, 
		       int nbProj, 
		       const int *d_projset, 
		       const double *d_pSrc, 
		       const double *d_rSrc, 
		       const double *d_rDet,   
		       double sizeDet, int pixDet, 
		       double spObj,
		       int row, int col, 
		       const double *d_X,
		       double *d_Y)
{
  /* Invoke Kernel */
  size_t GRIDSIZE = (size_t)ceil(sqrt(row * col));
  //printf("%d\n",GRIDSIZE);
  dim3 gridDim(GRIDSIZE, GRIDSIZE);

  //printf("radius = %f\n", radius);
  MARTOp_Kernel<<<gridDim, nbProj>>>(fanbeam,
				     d_projset,
				     d_pSrc,
				     d_rSrc,
				     d_rDet,
				     sizeDet, pixDet,
				     spObj,
				     row, col,
				     d_X,
				     d_Y);
  
  return hipGetLastError();
}



