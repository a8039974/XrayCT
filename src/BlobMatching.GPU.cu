#include "hip/hip_runtime.h"
/*  -*- C -*-  */
/* C interface for BlobMatching Kernel */
/* This file must be named .cu to be successfully compiled by nvcc */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "GPUTypes.hpp"
#include "BlobMatching.Kernel.cu"

hipError_t BlobMatching_GPU(BlobType blobtype,  /* The profile function identifier */
			     double radius, 
			     double mul, 
			     double dil, /* radius is that of dilated blob */
			     double alpha, double beta,			      
			     bool fanbeam, 
			     int nbProj, 
			     const int *d_projset, 
			     const double *d_pSrc, 
			     const double *d_rSrc, 
			     const double *d_rDet,   
			     double sizeDet, int pixDet, 
			     const double *d_T, size_t dimT,
			     double thetax, double thetay, /* Blob grid's generating vector, the corresponding angle : -pi/2 <= theta < 0*/
			     double splStep, /* sampling step of blob grid */
			     int row, int col, /* Blob grid dimension, even X even */   
			     const int *d_Idx,
			     int nbNode,
			     const double *d_X,
			     const bool *d_M,
			     const double *d_Y,
			     double *d_Z)
{
  /* Invoke Kernel */
  size_t GRIDSIZE = (size_t)ceil(sqrt(nbNode));
  //printf("%d\n",GRIDSIZE);
  dim3 gridDim(GRIDSIZE, GRIDSIZE);

  BlobMatching_Kernel<<<gridDim, nbProj>>>(blobtype,
					   radius,
					   mul,
					   dil,
					   alpha, beta,
					   fanbeam,
					   d_projset,
					   d_pSrc,
					   d_rSrc,
					   d_rDet,
					   sizeDet, pixDet,
					   d_T, dimT, 
					   thetax, thetay,
					   splStep,
					   row, col,
					   d_Idx,
					   nbNode,
					   d_X,
					   d_M,
					   d_Y,
					   d_Z);

  return hipGetLastError();
}
