#include "hip/hip_runtime.h"
/*  -*- C -*-  */
/* C interface for BlobProjector Kernel */
/* This file must be named .cu to be successfully compiled by nvcc */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "GPUTypes.hpp"
#include "BlobProjector.Kernel.cu"
#include "BlobProjector_StripIntegral.Kernel.cu"

hipError_t BlobProjector_GPU(BlobType blobtype,  /* The profile function identifier */
			      double radius, 
			      double mul, 
			      double dil, /* radius is that of dilated blob */
			      double alpha, double beta,			      
			      bool fanbeam, 
			      int nbProj, 
			      const int *d_projset, 
			      const double *d_pSrc, 
			      const double *d_rSrc, 
			      const double *d_rDet,   
			      double sizeDet, int pixDet, 
			      const double *d_T, size_t dimT, bool stripint, /* if stripint = true, d_T is treated as a strip-integral table, otherwise as direct footprint table. */
			      double thetax, double thetay, /* Blob grid's generating vector, the corresponding angle : -pi/2 <= theta < 0*/
			      double splStep, /* sampling step of blob grid */
			      int row, int col, /* Blob grid dimension, even X even */   
			      const int *d_Idx,
			      int nbNode,
			      const double *d_X,
			      const bool *d_MX,
			      double *d_Y,
			      const bool *d_MY,
			      bool forward)
{
  /* Invoke Kernel */
  size_t GRIDSIZE = (size_t)ceil(sqrt(nbNode));
  //printf("%d\n",GRIDSIZE);
  dim3 gridDim(GRIDSIZE, GRIDSIZE);

  if (stripint) {
    if (d_T != NULL) {
      BlobProjector_StripIntegral_Kernel<<<gridDim, nbProj>>>(blobtype,
							      radius,
							      mul,
							      dil,
							      alpha, beta,
							      fanbeam,
							      d_projset,
							      d_pSrc,
							      d_rSrc,
							      d_rDet,
							      sizeDet, pixDet,
							      d_T, dimT, 
							      thetax, thetay,
							      splStep,
							      row, col,
							      d_Idx,
							      nbNode,
							      d_X,
							      d_MX,
							      d_Y,
							      d_MY,
							      forward);    
    }
    else {
      printf("Strip-integral projector error: Table is emptpy!\n");
      exit(0);
    }
  }
  else  {
    BlobProjector_Kernel<<<gridDim, nbProj>>>(blobtype,
					      radius,
					      mul,
					      dil,
					      alpha, beta,
					      fanbeam,
					      d_projset,
					      d_pSrc,
					      d_rSrc,
					      d_rDet,
					      sizeDet, pixDet,
					      d_T, dimT, 
					      thetax, thetay,
					      splStep,
					      row, col,
					      d_Idx,
					      nbNode,
					      d_X,
					      d_MX,
					      d_Y,
					      d_MY,
					      forward);
  }

  return hipGetLastError();
}


